#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/native/cuda/SegmentReduceKernels.h>

#include <ATen/core/Tensor.h>
#include <ATen/Dispatch.h>
#include <ATen/NumericUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#else
#include <ATen/ops/zeros.h>
#include <ATen/ops/cat.h>
#include <ATen/ops/cumsum.h>
#endif

namespace at::native {

template <typename scalar_t, typename index_t>
__global__ void segment_reduce_backward_kernel(
    ReductionType reduction,
    scalar_t* grad_input_data,
    const scalar_t* grad_data,
    const scalar_t* output_data,
    const scalar_t* values_data,
    const index_t* lengths_data,
    const index_t* lengths_cumsum_data,
    const int64_t segment_count,
    const int64_t lengths_stride_axis,
    scalar_t initial_prod_value,
    const int64_t outer_offset,
    const int64_t inner_offset,
    const int64_t data_stride_axis,
    const int64_t data_size_axis,
    const int64_t output_stride_axis,
    const int64_t output_size_axis,
    const int64_t lengths_cumsum_stride_axis) {
  int64_t idx = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x;
  if (idx >= (outer_offset * segment_count * inner_offset)) {
    return;
  }
  int64_t row_id = idx / inner_offset;
  int64_t lane_id = idx % inner_offset;  // lane_id is the inner_idx
  int64_t outer_idx = row_id / segment_count;
  int64_t dim_idx = row_id % segment_count;

  int64_t lengths_idx = outer_idx * lengths_stride_axis * segment_count + dim_idx;
  auto segment_length = lengths_data[lengths_idx];
  if (segment_length == 0) {
    return;
  }

  int64_t offset_idx = outer_idx * lengths_cumsum_stride_axis * (segment_count + 1) + dim_idx;
  index_t offset_start = lengths_cumsum_data[offset_idx];
  index_t offset_end = lengths_cumsum_data[offset_idx + 1];

  int64_t output_index = outer_idx * output_stride_axis * output_size_axis
                         + dim_idx * output_stride_axis + lane_id;

  if (reduction == ReductionType::MAX ||
      reduction == ReductionType::MIN) {
    int64_t counter = 0;
    for (int64_t j = offset_start; j < offset_end; ++j) {
      int64_t data_index = outer_idx * data_stride_axis * data_size_axis
                           + j * data_stride_axis + lane_id;
      if (at::_isnan(values_data[data_index]) ||
          values_data[data_index] == output_data[output_index]) {
        grad_input_data[data_index] = grad_data[output_index];
        counter++;
      }
    }
    // Average gradient based on number of maximum elements in the
    // segment
    if (counter < 2) {
      return;
    }
    for (int64_t j = offset_start; j < offset_end; ++j) {
      int64_t data_index = outer_idx * data_stride_axis * data_size_axis
                           + j * data_stride_axis + lane_id;
      if (grad_input_data[data_index] > 0) {
        grad_input_data[data_index] =
            grad_input_data[data_index] / counter;
      }
    }
  } else if (reduction == ReductionType::MEAN) {
    auto grad_val = grad_data[output_index] / segment_length;
    for (int64_t j = offset_start; j < offset_end; ++j) {
      int64_t data_index = outer_idx * data_stride_axis * data_size_axis
                           + j * data_stride_axis + lane_id;
      grad_input_data[data_index] = grad_val;
    }
  } else if (reduction == ReductionType::SUM) {
    const auto& grad_val = grad_data[output_index];
    for (int64_t j = offset_start; j < offset_end; ++j) {
      int64_t data_index = outer_idx * data_stride_axis * data_size_axis
                           + j * data_stride_axis + lane_id;
      grad_input_data[data_index] = grad_val;
    }
  } else if (reduction == ReductionType::PROD) {
    const auto& grad_val = grad_data[output_index] * output_data[output_index];
    for (int64_t j = offset_start; j < offset_end; ++j) {
      int64_t data_index = outer_idx * data_stride_axis * data_size_axis
                           + j * data_stride_axis + lane_id;
      if (at::_isnan(values_data[data_index]) ||
          values_data[data_index] == 0) {
        // explicitly compute exclusive prod
        scalar_t exclusive_prod = initial_prod_value;
        int64_t prod_idx;
        for (int64_t k = offset_start; k < offset_end; ++k) {
          if (k != j) {
            prod_idx = outer_idx * data_stride_axis * data_size_axis
                       + k * data_stride_axis + lane_id;
            exclusive_prod *= values_data[prod_idx];
          }
        }
        grad_input_data[data_index] = grad_data[output_index] * exclusive_prod;
      } else {
        grad_input_data[data_index] = grad_val / values_data[data_index];
      }
    }
  }
}

Tensor _segment_reduce_lengths_offsets_backward_cuda_kernel(
    const Tensor& grad_contig,
    const Tensor& output_contig,
    const Tensor& data_contig,
    ReductionType reduction,
    const Tensor& lengths_or_offsets_contig,
    int64_t axis,
    const std::optional<Scalar>& initial,
    bool is_offsets_like) {
  axis = lengths_or_offsets_contig.dim() - 1;
  int64_t segment_count = is_offsets_like ?
                          lengths_or_offsets_contig.size(axis) - 1 :
                          lengths_or_offsets_contig.size(axis);
  int64_t lengths_stride_axis = lengths_or_offsets_contig.stride(axis);
  auto grad_input = at::zeros({data_contig.sizes()}, grad_contig.options());

  auto offsets = lengths_or_offsets_contig;
  auto lengths = lengths_or_offsets_contig;
  if (is_offsets_like) {
    lengths = lengths.diff();
  } else {
    auto zeros_shape = offsets.sizes().vec();
    zeros_shape[axis] = 1;
    offsets = at::cat({at::zeros(zeros_shape, offsets.options()), offsets}, axis);
    offsets.cumsum_(axis);
  }

  // outer_offset is the size of the outer dimensions of output (before axis)
  // inner_offset is the size of the inner dimensions of output (after axis)
  int64_t outer_offset = 1, inner_offset = 1;
  for (int64_t d = 0; d < axis; d++) {
    outer_offset *= output_contig.size(d);
  }
  for (int64_t d = axis + 1; d < output_contig.dim(); d++) {
    inner_offset *= output_contig.size(d);
  }

  constexpr int threads_per_block = 256;
  int64_t num_blocks = (outer_offset * inner_offset * segment_count + threads_per_block - 1) / threads_per_block;

  num_blocks = std::max(num_blocks, (int64_t)1);

  auto data_stride_axis = data_contig.stride(axis);
  auto data_size_axis = data_contig.size(axis);
  auto output_stride_axis = output_contig.stride(axis);
  auto output_size_axis = output_contig.size(axis);
  auto offsets_stride_axis = offsets.stride(axis);

  AT_DISPATCH_INDEX_TYPES(
      lengths_or_offsets_contig.scalar_type(), "_segment_reduce_cuda_lengths_offsets_backward_kernel1", ([&] {
        const auto* lengths_data = lengths.const_data_ptr<index_t>();
        auto* offsets_data = offsets.const_data_ptr<index_t>();

        // TODO: Switch to TensorIterator for better maintainablility and
        // readability
        AT_DISPATCH_FLOATING_TYPES_AND2(
            kBFloat16,
            kHalf,
            data_contig.scalar_type(),
            "_segment_reduce_cpu",
            ([&]() {
              auto* output_data = output_contig.const_data_ptr<scalar_t>();
              auto* grad_data = grad_contig.const_data_ptr<scalar_t>();
              auto* grad_input_data = grad_input.mutable_data_ptr<scalar_t>();
              const auto* values_data = data_contig.const_data_ptr<scalar_t>();

              scalar_t initial_prod_value;
              if (initial.has_value()) {
                initial_prod_value = initial.value().to<scalar_t>();
              } else {
                initial_prod_value = 1;
              }

              segment_reduce_backward_kernel<scalar_t>
                  <<<num_blocks,
                     threads_per_block,
                     0,
                     at::cuda::getCurrentCUDAStream()>>>(
                      reduction,
                      grad_input_data,
                      grad_data,
                      output_data,
                      values_data,
                      lengths_data,
                      offsets_data,
                      segment_count,
                      lengths_stride_axis,
                      initial_prod_value,
                      outer_offset,
                      inner_offset,
                      data_stride_axis,
                      data_size_axis,
                      output_stride_axis,
                      output_size_axis,
                      offsets_stride_axis
                    );
              C10_CUDA_KERNEL_LAUNCH_CHECK();
            }));
      }));
  return grad_input;
}

Tensor _segment_reduce_lengths_backward_cuda_kernel(
  const Tensor& grad_contig,
  const Tensor& output_contig,
  const Tensor& data_contig,
  ReductionType reduction,
  const Tensor& lengths_contig,
  int64_t axis,
  const std::optional<Scalar>& initial) {
  return _segment_reduce_lengths_offsets_backward_cuda_kernel(
    grad_contig, output_contig, data_contig, reduction, lengths_contig, axis, initial, /*is_offsets_like=*/false);
}

Tensor _segment_reduce_offsets_backward_cuda_kernel(
  const Tensor& grad_contig,
  const Tensor& output_contig,
  const Tensor& data_contig,
  ReductionType reduction,
  const Tensor& offsets_contig,
  int64_t axis,
  const std::optional<Scalar>& initial) {
  return _segment_reduce_lengths_offsets_backward_cuda_kernel(
    grad_contig, output_contig, data_contig, reduction, offsets_contig, axis, initial, /*is_offsets_like=*/true);
}

} // namespace at::native 