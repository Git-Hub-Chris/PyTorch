#include "hip/hip_runtime.h"
#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/ceil_div.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/MemoryOverlap.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/native/cuda/MemoryAccess.cuh>
#include <ATen/native/Resize.h>
#include <ATen/native/TypeProperties.h>
#include <ATen/native/TensorShape.h>
#include <ATen/Dispatch.h>
#include <ATen/Dispatch_v2.h>
#include <c10/core/MemoryFormat.h>

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
#include <cuda/barrier>
#include <cuda/ptx>
#endif

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/cat_native.h>
#include <ATen/ops/copy_native.h>
#include <ATen/ops/empty.h>
#include <ATen/ops/empty_like.h>
#include <ATen/ops/narrow.h>
#endif

namespace at::native {

constexpr int CAT_ARRAY_BATCH_SIZE = 128;
constexpr int CAT_ARRAY_MAX_INPUT_DIMS = 4;
constexpr int ALIGNED_VEC_LOAD_BYTES = 16;
constexpr int ALIGNED_COLUMN_COUNT = 4;
constexpr int SMEM_BUFFER_SIZE_BYTES = 16384;
constexpr int NUM_TMA_STAGE = 2;
constexpr int SMEM_SIZE = 49152;

namespace {

inline bool is_aligned_vec4(const void* ptr) {
  auto iptr = reinterpret_cast<uintptr_t>(ptr);
  return !(iptr % alignof(int4));
}

inline bool getCatGrid(ptrdiff_t nTensors, dim3& grid) {
  const int numSM = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

  // X dim of grid for cat array cooperates on a single tensor in the cat.
  // Given half of the GPU, full utilization will always occur.

  // This will have cating two tensors fill the entire grid, but prevent
  // many threads from needlessly load meta data if their sizes is small.

  grid = dim3( 2LL * numSM, (long long) nTensors );

  return true;
}

template<typename T>
inline std::tuple<dim3, dim3> getCatGridRocm(unsigned int max_elements_per_tensor,
  ptrdiff_t nTensors) {
  constexpr unsigned int threads_per_block = 256;
  constexpr unsigned int elements_per_thread = 8;
  constexpr unsigned int max_tb_per_sm = 32;

  unsigned int max_threads = ceil_div(max_elements_per_tensor, elements_per_thread);
  unsigned int thread_blocks = ceil_div(max_threads, threads_per_block);

  // Limit the number of thread blocks to prevent too many threads to load the metadata
  // if they operate on very small tensors.

  const unsigned int num_sm = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  thread_blocks = std::min(num_sm * max_tb_per_sm, thread_blocks);

  dim3 block = dim3(threads_per_block);
  dim3 grid = dim3(thread_blocks, (long long)nTensors);

  return std::make_tuple(grid, block);
}

template<typename T>
inline std::tuple<dim3, dim3> getCatGridContig(unsigned int max_elements_per_tensor,
  ptrdiff_t nTensors, bool use_fast_tma) {

  unsigned int threads_per_block =  128;
  if(use_fast_tma)
    threads_per_block =  512;

  constexpr unsigned int min_aligned_vec_per_thread = 1;
  constexpr unsigned int max_tb_per_sm = 32;

  unsigned int elements_per_thread = ALIGNED_VEC_LOAD_BYTES / sizeof(T) *
    min_aligned_vec_per_thread;
  unsigned int max_threads = ceil_div(max_elements_per_tensor, elements_per_thread);
  unsigned int thread_blocks = ceil_div(max_threads, threads_per_block);

  // Limit the number of thread blocks to prevent too many threads to load the metadata
  // if they operate on very small tensors.

  const unsigned int num_sm = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  thread_blocks = std::min(num_sm * max_tb_per_sm, thread_blocks);

  dim3 block = dim3(threads_per_block);
  dim3 grid = dim3(thread_blocks, (long long)nTensors);

  return std::make_tuple(grid, block);
}

// Similar to any other IndexToOffset calculation for copying along a given
// dimension.
template <typename IndexType, int Dims>
struct CatArrIndexToOffset {
  static inline __device__ IndexType compute(
      const IndexType tensorSize[Dims],
      const IndexType tensorStride[Dims],
      const IndexType dimSize,
      const unsigned int concatDim,
      IndexType linearIndex) {
    // linearIndex is not really linear index, but instead the offset in
    // input tensor. If the input tensor is contiguous, then this offset
    // is the linear index, but if the input tensor is channels last, then
    // it is the linear index of the permuted contiguous tensor
    IndexType offset = 0;

    #pragma unroll
    for (int i = Dims - 1; i >= 1; --i) {
      IndexType curDimSize = i == concatDim ? dimSize : tensorSize[i];
      IndexType nextDimIndex = linearIndex / curDimSize;
      IndexType curDimIndex = linearIndex - curDimSize * nextDimIndex;
      IndexType curDimOffset = curDimIndex * tensorStride[i];
      offset += curDimOffset;
      linearIndex = nextDimIndex;
    }

    return offset + linearIndex * tensorStride[0];
  }
};

template<typename IndexType, unsigned int MaxDims>
struct TensorSizeStride {
  IndexType tensorSize[MaxDims];
  IndexType tensorStride[MaxDims];
};

/**
  * Kernel used to concatenated grimDim.y tensors into an output tensor. Uses a
  * grid-stride loop based off of the blockIdx.x, threadIdx.x for each input to
  * copy each element from each input tensor into the output.
  *
  * output: base pointer to the storage associated with the output tensor
  * inputs: GPU-allocated array of input metadata for each input to concatenate
  *         in the kernel
  * os: the size/stride vectors for the output tensor
  * concatDim: dimension along which we are concatenating
  * dimStride: the stride of the output tensor at the concatDim
  *
  * The most important assumption made is that the input tensors are contiguous.
  */


// pass meta data directly through kernel argument instead of pin memory
// In contiguous case, we will not need stride_size, setting it as 1 as placeholder
// to pass compile.
template <typename T, typename IndexType, int n, int stride_size>
struct CatArrInputTensorMetadata {
  const T* input[n];
  IndexType offset[n];
  IndexType dimSize[n];
  IndexType nElements[n];
  bool isContiguous[n];
  TensorSizeStride<IndexType, CAT_ARRAY_MAX_INPUT_DIMS> tensorStride[stride_size];
};

template <typename T, typename IndexType, int Dims, int batch_size, int stride_size>
__global__ void CatArrayBatchedCopy(
    T* output,
    CatArrInputTensorMetadata<T, IndexType, batch_size, stride_size> inputs,
    TensorSizeStride<IndexType, CAT_ARRAY_MAX_INPUT_DIMS> os,
    const int concatDim,
    IndexType dimStride) {

    IndexType tid = blockIdx.x * blockDim.x + threadIdx.x;
    IndexType nElements = inputs.nElements[blockIdx.y];
    TensorSizeStride<IndexType, CAT_ARRAY_MAX_INPUT_DIMS> ins = stride_size > 1 ? inputs.tensorStride[blockIdx.y] : inputs.tensorStride[0];
    bool isContig = inputs.isContiguous[blockIdx.y];

    if(tid >= nElements) return;

    const T* data = inputs.input[blockIdx.y];
    IndexType offset = inputs.offset[blockIdx.y];
    IndexType dimSize = inputs.dimSize[blockIdx.y];
    IndexType dataOffset = offset * dimStride;

    IndexType stride = gridDim.x * blockDim.x;

    while( tid < nElements){
      IndexType elementOffset = CatArrIndexToOffset<IndexType, Dims>::compute(
                    os.tensorSize, os.tensorStride, dimSize, concatDim, tid);
      if (isContig) {
        output[dataOffset + elementOffset] = data[tid];
      } else {
        IndexType inElementOffset = CatArrIndexToOffset<IndexType, Dims>::compute(
                    ins.tensorSize, ins.tensorStride, dimSize, concatDim, tid);
        output[dataOffset + elementOffset] = data[inElementOffset];
      }
    tid += stride;
    }
}

template <typename T, typename IndexType, int Dims, int batch_size, int stride_size>
__global__ void CatArrayBatchedCopy_contig(
    T* output,
    CatArrInputTensorMetadata<T, IndexType, batch_size, stride_size> inputs,
    TensorSizeStride<IndexType, CAT_ARRAY_MAX_INPUT_DIMS> os,
    const int concatDim,
    IndexType dimStride) {

    IndexType tid = blockIdx.x * blockDim.x + threadIdx.x;
    IndexType nElements = inputs.nElements[blockIdx.y];

    if(tid >= nElements) return;

    const T* data = inputs.input[blockIdx.y];
    IndexType offset = inputs.offset[blockIdx.y];
    IndexType dimSize = inputs.dimSize[blockIdx.y];
    IndexType dataOffset = offset * dimStride;

    IndexType stride = gridDim.x * blockDim.x;

    while( tid < nElements){
      IndexType elementOffset = CatArrIndexToOffset<IndexType, Dims>::compute(
                    os.tensorSize, os.tensorStride, dimSize, concatDim, tid);
      output[dataOffset + elementOffset] = data[tid];
      tid += stride;
    }
}

/*
  Specialized implementation of the CatArrayBatchedCopy written to generate wide memory loads
  to improve memory bandwidth throughput.
*/

template <typename T, typename IndexType, int Dims, int batch_size, int stride_size>
__global__ void CatArrayBatchedCopy_aligned16_contig(
    T* output,
    CatArrInputTensorMetadata<T, IndexType, batch_size, stride_size> inputs,
    TensorSizeStride<IndexType, CAT_ARRAY_MAX_INPUT_DIMS> os,
    const int concatDim,
    IndexType dimStride) {

    // This kernel tries to use 128 bit loads
    constexpr int kILP = ALIGNED_VEC_LOAD_BYTES / sizeof(T);
    IndexType inputOffset = (blockIdx.x * blockDim.x + threadIdx.x) * kILP;
    IndexType inputStride = gridDim.x * blockDim.x * kILP;

    IndexType nElements = inputs.nElements[blockIdx.y];
    if (inputOffset >= nElements) {
      return;
    }

    const T* data = inputs.input[blockIdx.y];
    IndexType offset = inputs.offset[blockIdx.y];
    IndexType dimSize = inputs.dimSize[blockIdx.y];
    IndexType dataOffset = offset * dimStride;

    IndexType v_elementOffset[kILP];
    T reg_data[kILP];

    while (inputOffset + kILP <= nElements) {
      for (int i = 0; i < kILP; ++i) {
        v_elementOffset[i] = CatArrIndexToOffset<IndexType, Dims>::compute(os.tensorSize,
          os.tensorStride, dimSize, concatDim, inputOffset + i);
      }

      using LT = at::native::memory::aligned_vector<T, kILP>;
      ((LT*)reg_data)[0] = const_cast<LT*>((LT*)(data + inputOffset))[0];

      #pragma unroll
      for (int i = 0; i < kILP; ++i) {
        output[dataOffset + v_elementOffset[i]] = reg_data[i];
      }

      inputOffset += inputStride;
    }

    // Handle remaining tail in case nElements does not divide
    // exactly to kILP

    while (inputOffset < nElements) {
      v_elementOffset[0] = CatArrIndexToOffset<IndexType, Dims>::compute(os.tensorSize,
        os.tensorStride, dimSize, concatDim, inputOffset);
      output[dataOffset + v_elementOffset[0]] = data[inputOffset];
      inputOffset++;
    }
}

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
template <int block_dim, int stage_num>
struct SMEM {
  alignas(16) char in_stage[stage_num][block_dim];
  alignas(16) ::cuda::barrier<::cuda::thread_scope_block> bar[stage_num];
};

template <typename = void>
__device__ static inline bool elect_sync(const std::uint32_t& membermask) {
  std::uint32_t is_elected;
  asm volatile(
    "{\n\t .reg .pred P_OUT; \n\t"
    "elect.sync _|P_OUT, %1;\n\t"
    "selp.b32 %0, 1, 0, P_OUT; \n"
    "}"
    : "=r"(is_elected)
    : "r"(membermask)
    :);
  return static_cast<bool>(is_elected);
}

template <typename T, typename ITYPE, typename IndexType, int inputsNum>
struct Parameters {
  int lengths[inputsNum];
  const T* data[inputsNum];
  IndexType nElements[inputsNum];
  int buffer_len = 0;
  int Dims[inputsNum];
  IndexType dimSize_total = 0;
  IndexType src_start[inputsNum];
  uint16_t f2D = 0;
  uint16_t each_length[inputsNum];
  IndexType total_len = 0;
  uint8_t repeats = 0;
};

template <typename T, typename ITYPE, typename IndexType, int batch_size, int stride_size, int inputsNum, int block_dim>
__device__ void initial_parameters(
  CatArrInputTensorMetadata<T, IndexType, batch_size, stride_size> inputs,
  Parameters<T, ITYPE, IndexType, inputsNum> &cal_parameters,
  int scalar_t_num){
  #pragma unroll
  for(int i=0; i<inputsNum; i++){
    cal_parameters.dimSize_total += inputs.dimSize[i];
    cal_parameters.nElements[i] = inputs.nElements[i]/scalar_t_num;
    cal_parameters.Dims[i] = inputs.dimSize[i]/scalar_t_num;
    cal_parameters.data[i] = reinterpret_cast<const T*>(inputs.input[i]);
    cal_parameters.f2D += cal_parameters.Dims[i];
    cal_parameters.total_len += cal_parameters.nElements[i];
  }

  cal_parameters.dimSize_total /= scalar_t_num;
  cal_parameters.repeats = blockDim.x / cal_parameters.dimSize_total;
  int num_loads = block_dim / sizeof(ITYPE) / cal_parameters.dimSize_total;
  cal_parameters.f2D -= cal_parameters.Dims[inputsNum - 1];

  #pragma unroll
  for(int i=0; i<inputsNum; i++){
    cal_parameters.lengths[i] = min(num_loads * cal_parameters.Dims[i], cal_parameters.nElements[i]);
    cal_parameters.buffer_len += cal_parameters.lengths[i];
    cal_parameters.each_length[i] = cal_parameters.repeats * cal_parameters.Dims[i];
  }
}

using ArrivalToken = ::cuda::barrier<::cuda::thread_scope_block>::arrival_token;

template <typename T, typename ITYPE, typename IndexType, int inputsNum, int block_dim, int num_stages>
__device__ void TMA_load(
  int stage,
  int scalar_t_num,
  Parameters<T, ITYPE, IndexType, inputsNum> &cal_parameters,
  SMEM<block_dim, num_stages> &smem,
  ArrivalToken (&token)[num_stages]){

  int acc_length = 0, acc_length_bytes = 0;
  if(cal_parameters.src_start[0] < cal_parameters.nElements[0]){
    #pragma unroll
    for(int i = 0; i < inputsNum; i++){
      IndexType length = cal_parameters.src_start[i] + cal_parameters.lengths[i] < cal_parameters.nElements[i] ? cal_parameters.lengths[i] : cal_parameters.nElements[i] - cal_parameters.src_start[i];
      IndexType length_bytes = length * sizeof(ITYPE);
      ::cuda::ptx::cp_async_bulk(
            ::cuda::ptx::space_cluster,
            ::cuda::ptx::space_global,
            &smem.in_stage[stage][acc_length],
            &cal_parameters.data[i][cal_parameters.src_start[i]*scalar_t_num],
            length_bytes,
            (uint64_t*)&smem.bar[stage]);
      acc_length += cal_parameters.lengths[i] * sizeof(ITYPE);
      acc_length_bytes += length_bytes;
    }
    token[stage]= ::cuda::ptx::mbarrier_arrive_expect_tx(
      ::cuda::ptx::sem_release,
      ::cuda::ptx::scope_cta,
      ::cuda::ptx::space_shared,
      (uint64_t*)&smem.bar[stage],
      acc_length_bytes);
  }
}

template<typename T, typename ITYPE, typename IndexType, int inputsNum>
__device__ uint16_t cal_index(
  Parameters<T, ITYPE, IndexType, inputsNum> cal_parameters,
  uint16_t mod,
  uint16_t pos_r){
  uint16_t in_index;
  if(inputsNum == 2)
    in_index = mod < cal_parameters.Dims[0] ? (pos_r*cal_parameters.Dims[0] + mod) : (cal_parameters.lengths[0] + pos_r * cal_parameters.Dims[1] + mod - cal_parameters.Dims[0]);
  else if (inputsNum == 3)
    in_index = mod < cal_parameters.Dims[0] ? (pos_r*cal_parameters.Dims[0] + mod) : ((mod < cal_parameters.f2D) ? (cal_parameters.lengths[0] + pos_r * cal_parameters.Dims[1] + mod - cal_parameters.Dims[0]) : (cal_parameters.lengths[0] + cal_parameters.lengths[1] + pos_r * cal_parameters.Dims[2] + mod - cal_parameters.f2D));
  return in_index;
}
#endif

//TMA - generic kernel
template <typename T, typename IndexType, int Dims, int batch_size, int stride_size, int num_stages=1, int smem_bytes=128>
__global__ void CatArrayBatchedCopy_contig_TMA(
    T* output,
    CatArrInputTensorMetadata<T, IndexType, batch_size, stride_size> inputs,
    TensorSizeStride<IndexType, CAT_ARRAY_MAX_INPUT_DIMS> os,
    const int concatDim,
    IndexType dimStride) {

    #if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
    IndexType tid = blockIdx.x * blockDim.x + threadIdx.x;
    IndexType nElements = inputs.nElements[blockIdx.y];

    using ITYPE = at::native::memory::aligned_vector<T, ALIGNED_COLUMN_COUNT>;
    int scalar_t_num = ALIGNED_COLUMN_COUNT;

    if(tid >= nElements) return;
    IndexType offset = inputs.offset[blockIdx.y];
    IndexType dimSize = inputs.dimSize[blockIdx.y];
    IndexType dataOffset = offset * dimStride / scalar_t_num;

    extern __shared__ char dynamic_smem[];

    using SMEM_T = SMEM<smem_bytes, num_stages>;
    IndexType block_dim = smem_bytes/sizeof(ITYPE);
    IndexType smem_stride = gridDim.x * block_dim;

    SMEM_T &smem = reinterpret_cast<SMEM_T&>(dynamic_smem);
    ITYPE *output_vec = reinterpret_cast<ITYPE*>(output);

    ::cuda::barrier<::cuda::thread_scope_block>::arrival_token token[num_stages];

    const ITYPE* data = reinterpret_cast<const ITYPE*>(inputs.input[blockIdx.y]);
    IndexType h2_nElements = nElements/scalar_t_num;
    bool elected = elect_sync(~0);

    if (threadIdx.x < 32 && elected) {
      #pragma unroll
      for (int stage = 0; stage < num_stages; ++stage) {
        IndexType src_start = smem_stride * stage + blockIdx.x * block_dim;
        if(src_start < h2_nElements){
          IndexType length = src_start + block_dim < h2_nElements ? block_dim : h2_nElements - src_start;
          const ITYPE* src = &data[src_start];
          IndexType length_bytes = length * sizeof(ITYPE);

          init(&smem.bar[stage], 1);
          ::cuda::ptx::fence_proxy_async(::cuda::ptx::space_shared);
          ::cuda::ptx::cp_async_bulk(
                  ::cuda::ptx::space_cluster,
                  ::cuda::ptx::space_global,
                  &smem.in_stage[stage],
                  src,
                  length_bytes,
                  (uint64_t*)&smem.bar[stage]);
          token[stage] = ::cuda::ptx::mbarrier_arrive_expect_tx(
                  ::cuda::ptx::sem_release,
                  ::cuda::ptx::scope_cta,
                  ::cuda::ptx::space_shared,
                  (uint64_t*)&smem.bar[stage],
                  length_bytes);
        }
      }
    }
    int stage = 0;
    for (int idx = (blockIdx.x * block_dim); idx < h2_nElements; idx += smem_stride) {
      bool elected = elect_sync(~0);
      if(threadIdx.x < 32 && elected){
        smem.bar[stage].wait(::cuda::std::move(token[stage]));
      }
      __syncthreads();

      IndexType idx_global = idx + threadIdx.x;
      ITYPE *smem_vec = reinterpret_cast<ITYPE*>(smem.in_stage[stage]);
      for(int out_num = 0; out_num < block_dim; out_num += blockDim.x){
        if (idx_global < h2_nElements) {
          IndexType elementOffset = CatArrIndexToOffset<IndexType, Dims>::compute(os.tensorSize,
                                                        os.tensorStride, dimSize, concatDim, idx_global * scalar_t_num);

          output_vec[dataOffset + elementOffset/scalar_t_num] = smem_vec[out_num + threadIdx.x];
        }
        idx_global += blockDim.x;
      }
      IndexType src_start = smem_stride * num_stages + idx;
      __syncthreads();

      elected = elect_sync(~0);
      if(threadIdx.x < 32 && elected && src_start < h2_nElements){
        IndexType length = src_start + block_dim < h2_nElements ? block_dim : h2_nElements - src_start;
        const ITYPE* src = &data[src_start];
        IndexType length_bytes = length * sizeof(ITYPE);
        ::cuda::ptx::cp_async_bulk(
                ::cuda::ptx::space_cluster,
                ::cuda::ptx::space_global,
                &smem.in_stage[stage],
                src,
                length_bytes,
                (uint64_t*)&smem.bar[stage]);
        token[stage] = ::cuda::ptx::mbarrier_arrive_expect_tx(
                ::cuda::ptx::sem_release,
                ::cuda::ptx::scope_cta,
                ::cuda::ptx::space_shared,
                (uint64_t*)&smem.bar[stage],
                length_bytes);
      }
      stage = (stage + 1) % num_stages;
  }
  #endif
}

//TMA specialized kernel for 2 and 3 input cases
//It could have around 2x speedup vs the general version on bf16 datatype
//It will only be used when the output column dim smaller than the shared memory buffer size
template <typename T, typename IndexType, int Dims, int batch_size, int stride_size, int inputsNum = 2, int num_stages=1, int smem_bytes=128>
__global__ void CatArrayBatchedCopy_contig_TMA_fast(
    T* output,
    CatArrInputTensorMetadata<T, IndexType, batch_size, stride_size> inputs,
    TensorSizeStride<IndexType, CAT_ARRAY_MAX_INPUT_DIMS> os,
    const int concatDim,
    IndexType dimStride) {
    #if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
    IndexType tid = blockIdx.x * blockDim.x + threadIdx.x;
    IndexType nElements = inputs.nElements[blockIdx.y];

    using ITYPE = at::native::memory::aligned_vector<T, ALIGNED_COLUMN_COUNT>;
    int scalar_t_num = ALIGNED_COLUMN_COUNT;

    extern __shared__ char dynamic_smem[];

    using SMEM_T = SMEM<smem_bytes, num_stages>;
    SMEM_T &smem = reinterpret_cast<SMEM_T&>(dynamic_smem);
    ITYPE *output_vec = reinterpret_cast<ITYPE*>(output);

    Parameters<T, ITYPE, IndexType, inputsNum> cal_parameters;
    initial_parameters<T, ITYPE, IndexType, batch_size, stride_size, inputsNum, smem_bytes>(inputs, cal_parameters, scalar_t_num);

    ArrivalToken token[num_stages];

    uint16_t wok_threads = cal_parameters.repeats * cal_parameters.dimSize_total;
    uint16_t pos_r = threadIdx.x / cal_parameters.dimSize_total;
    uint16_t mod = threadIdx.x % cal_parameters.dimSize_total;
    IndexType smem_stride = gridDim.x * cal_parameters.buffer_len;

    bool elected = elect_sync(~0);
    if (threadIdx.x < 32 && elected) {
      #pragma unroll
      for (int stage = 0; stage < num_stages; ++stage) {
        init(&smem.bar[stage], 1);
        ::cuda::ptx::fence_proxy_async(::cuda::ptx::space_shared);
        #pragma unroll
        for(int i = 0; i < inputsNum; i++){
          cal_parameters.src_start[i] = gridDim.x * cal_parameters.lengths[i] * stage + blockIdx.x * cal_parameters.lengths[i];
        }
        TMA_load<T, ITYPE, IndexType, inputsNum, smem_bytes, num_stages>(stage, scalar_t_num, cal_parameters, smem, token);
      }
    }
    int stage = 0;

    #pragma unroll
    for(int i=0; i<inputsNum; i++){
      cal_parameters.src_start[i] = gridDim.x * cal_parameters.lengths[i] * (num_stages - 1) + blockIdx.x * cal_parameters.lengths[i];
    }

    uint16_t in_index = cal_index<T, ITYPE, IndexType, inputsNum>(cal_parameters, mod, pos_r);
    IndexType idx_global = blockIdx.x * cal_parameters.buffer_len;

    while(idx_global < cal_parameters.total_len) {
      bool elected = elect_sync(~0);
      if(threadIdx.x < 32 && elected){
        smem.bar[stage].wait(::cuda::std::move(token[stage]));
      }
      __syncthreads();

      uint16_t tmp_index = in_index;
      ITYPE *smem_vec = reinterpret_cast<ITYPE*>(smem.in_stage[stage]);
      #pragma unroll
      for(int tid = threadIdx.x; tid < cal_parameters.buffer_len; tid += wok_threads){
        if (threadIdx.x < wok_threads && idx_global +tid < cal_parameters.total_len) {
          output_vec[idx_global + tid] = smem_vec[tmp_index];
          if(inputsNum == 2){
            tmp_index += mod < cal_parameters.Dims[0] ? cal_parameters.each_length[0] : cal_parameters.each_length[1];
          }else if(inputsNum == 3){
            tmp_index += mod < cal_parameters.Dims[0] ? cal_parameters.each_length[0] : ((mod < cal_parameters.f2D) ? cal_parameters.each_length[1] : cal_parameters.each_length[2]);
          }
        }
      }
      idx_global += smem_stride;
      __syncthreads();

      #pragma unroll
      for(int i=0; i<inputsNum; i++){
        cal_parameters.src_start[i] += gridDim.x * cal_parameters.lengths[i];
      }
      elected = elect_sync(~0);
      if(threadIdx.x < 32 && elected){
        TMA_load<T, ITYPE, IndexType, inputsNum, smem_bytes, num_stages>(stage, scalar_t_num, cal_parameters, smem, token);
      }
      stage = (stage + 1) % num_stages;
  }
  #endif
}

template <typename scalar_t, int batch_size, int stride_size>
void parallel_cat(const Tensor &out, const MaterializedITensorListRef& inputs, int64_t dimension,
                  int nDims, c10::MemoryFormat memory_format) {
  // First, let's set up our kernel parameters. We start with a raw pointer to
  // the storage for the output Tensor.
  scalar_t *data = (scalar_t *)(out.mutable_data_ptr());
  bool in_outIsAligned = is_aligned_vec4(data);
  CatArrInputTensorMetadata<scalar_t, unsigned int, batch_size, stride_size> catMetaData;
  TensorSizeStride<unsigned int, CAT_ARRAY_MAX_INPUT_DIMS> outputParam;

  // Next, let's initialize the size, stride arrays for the output Tensor.
  if (memory_format == c10::MemoryFormat::Contiguous) {
    for (int i = 0; i < nDims; ++i) {
      outputParam.tensorSize[i] = out.size(i);
      outputParam.tensorStride[i] = out.stride(i);
    }
  } else if (memory_format == c10::MemoryFormat::ChannelsLast || memory_format == c10::MemoryFormat::ChannelsLast3d) {
    // permute the semantics of dims from NCHW to NHWC so that the input
    // tensor is now contiguous
    outputParam.tensorSize[0] = out.size(0);
    outputParam.tensorStride[0] = out.stride(0);
    for (int i = 1; i < nDims - 1; ++i) {
      outputParam.tensorSize[i] = out.size(i + 1);
      outputParam.tensorStride[i] = out.stride(i + 1);
    }
    outputParam.tensorSize[nDims - 1] = out.size(1);
    outputParam.tensorStride[nDims - 1] = out.stride(1);
  } else {
    TORCH_CHECK(false, "unsupported memory format");
  }

  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

  // If all batches are contiguous we can call a specialized implementation
  // which requires the input tensor addresses to be aligned to a
  // 16 Byte boundary.

  bool isContig = true;
  bool isAligned = true;
  unsigned int max_elements_per_tensor = 0;

  // Now we loop
  int batchCounter = 0;
  int64_t offset = 0;
  for (unsigned i = 0; i < inputs.size() ; i += batch_size) {
    int64_t total_dims = 0;
    for (batchCounter = 0;
          batchCounter < batch_size &&
            (i+batchCounter) < inputs.size();
          ++batchCounter) {
      int64_t dimSize = 0;
      // There is a legacy case where a 1-D empty tensor can be concat with
      // high-dimensional tensor
      if (inputs[i+batchCounter].get().numel() > 0) {
        dimSize = inputs[i+batchCounter].get().size(dimension);
      }
      if(dimSize % ALIGNED_COLUMN_COUNT != 0){
        in_outIsAligned = false;
      }
      catMetaData.input[batchCounter] = (scalar_t*)(inputs[i+batchCounter].get().const_data_ptr());
      catMetaData.offset[batchCounter] = offset;
      catMetaData.dimSize[batchCounter] = dimSize;
      catMetaData.nElements[batchCounter] = inputs[i+batchCounter].get().numel();
      total_dims += dimSize;
#ifdef USE_ROCM
      // On ROCm, CatArrayBatchedCopy_contig is faster
      isAligned = false;
#else
      // If at least one of the inputs is not aligned, we can't call the
      // CatArrayBatchedCopy_aligned16_contig
      isAligned &= is_aligned_vec4(catMetaData.input[batchCounter]);
#endif

      if (stride_size > 1) {
        auto strides = inputs[i+batchCounter].get().strides();
        auto sizes = inputs[i+batchCounter].get().sizes();
        for(int j = 0; j < nDims; j++){
          catMetaData.tensorStride[batchCounter].tensorSize[j] = sizes[j];
          catMetaData.tensorStride[batchCounter].tensorStride[j] = strides[j];
        }
        catMetaData.isContiguous[batchCounter] = false;
        isContig = false;
      } else {
        catMetaData.isContiguous[batchCounter] = true;
      }

      // Update offset
      offset += dimSize;

      // We need max elements per tensor to compute grid parameters
      max_elements_per_tensor = std::max(max_elements_per_tensor,
        catMetaData.nElements[batchCounter]);
    }

    // Skip if the tensor is empty. Otherwise, the grid dim is invalid
    if (max_elements_per_tensor == 0)
      continue;

    dim3 applyBlock, catGrid;
    const int prop_major = at::cuda::getCurrentDeviceProperties()->major;
    bool use_tma = prop_major >= 9 && isContig && isAligned && in_outIsAligned && sizeof(scalar_t) >= 2;
    //The swizzle version will only be used when the shared memory buffer could hold at least 1 row of all the concat tensors
    bool use_fast_tma = use_tma && batchCounter > 1 && batchCounter < 4 && SMEM_BUFFER_SIZE_BYTES/ALIGNED_COLUMN_COUNT/sizeof(scalar_t) >= total_dims;

#ifdef USE_ROCM
    // always base grid size on max_elements_per_tensor
    auto [catGrid, applyBlock] = getCatGridRocm<scalar_t>(
          max_elements_per_tensor, batchCounter);
#else
  if(use_fast_tma){
    std::tie(catGrid, applyBlock) = getCatGridContig<scalar_t>(
      max_elements_per_tensor, 1, use_fast_tma);
  }else if (isContig && sizeof(scalar_t) > 2) {
    std::tie(catGrid, applyBlock) = getCatGridContig<scalar_t>(
        max_elements_per_tensor, batchCounter, use_fast_tma);
  }
  else {
    applyBlock = dim3(32 * 16);
    getCatGrid(batchCounter, catGrid);
  }
#endif

    if (memory_format != c10::MemoryFormat::Contiguous) {
      switch (dimension) {
      case 0:
        break;
      case 1:
        dimension = nDims - dimension;
        break;
      default:
        dimension--;
      }
    }

    // Template Declarations for dim = 1, 2, 3, 4
  #define HANDLE_CASE(DIMS) \
    if (use_fast_tma && batchCounter == 2) {\
      CatArrayBatchedCopy_contig_TMA_fast<scalar_t, unsigned int, DIMS, batch_size, stride_size, 2, NUM_TMA_STAGE, SMEM_BUFFER_SIZE_BYTES><<<\
      catGrid, applyBlock, SMEM_SIZE, stream.stream()>>>(\
              data, catMetaData, outputParam, dimension, outputParam.tensorStride[dimension]);\
    } else if (use_fast_tma && batchCounter == 3) {\
      CatArrayBatchedCopy_contig_TMA_fast<scalar_t, unsigned int, DIMS, batch_size, stride_size, 3, NUM_TMA_STAGE, SMEM_BUFFER_SIZE_BYTES><<<\
      catGrid, applyBlock, SMEM_SIZE, stream.stream()>>>(\
              data, catMetaData, outputParam, dimension, outputParam.tensorStride[dimension]);\
    } else if (use_tma) {\
      CatArrayBatchedCopy_contig_TMA<scalar_t, unsigned int, DIMS, batch_size, stride_size, NUM_TMA_STAGE, SMEM_BUFFER_SIZE_BYTES><<<\
      catGrid, applyBlock, SMEM_SIZE, stream.stream()>>>(\
              data, catMetaData, outputParam, dimension, outputParam.tensorStride[dimension]);\
    } else if (isContig && isAligned && sizeof(scalar_t) >= 4 && sizeof(scalar_t) <= 8) {\
      CatArrayBatchedCopy_aligned16_contig<scalar_t, unsigned int, DIMS, batch_size, stride_size><<<\
          catGrid, applyBlock, 0, stream.stream()>>>(\
              data, catMetaData, outputParam, dimension, outputParam.tensorStride[dimension]);\
    } else if (isContig) {\
      CatArrayBatchedCopy_contig<scalar_t, unsigned int, DIMS, batch_size, stride_size><<<\
          catGrid, applyBlock, 0, stream.stream()>>>(\
              data, catMetaData, outputParam, dimension, outputParam.tensorStride[dimension]);\
    } else {\
      CatArrayBatchedCopy<scalar_t, unsigned int, DIMS, batch_size, stride_size><<<\
          catGrid, applyBlock, 0, stream.stream()>>>(\
              data, catMetaData, outputParam, dimension, outputParam.tensorStride[dimension]);\
    }\
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    switch (nDims) {
      case 1:
        HANDLE_CASE(1);
        break;
      case 2:
        HANDLE_CASE(2);
        break;
      case 3:
        HANDLE_CASE(3);
        break;
      case 4:
        HANDLE_CASE(4);
        break;
    }
#undef HANDLE_CASE

  }
}
// The kernels are templated on an opaque, self-aligned type of the correct
// size to avoid redundant kernels for different types of the same size.
template <unsigned N> struct alignas(N) OpaqueType { char data[N]; };

} // namespace

TORCH_IMPL_FUNC(cat_out_cuda)
(const ITensorListRef& tensors,
 int64_t dim,
 int64_t valid,
 bool all_contiguous,
 bool all_same_dtype,
 bool all_same_sizes_and_stride,
 MemoryFormat memory_format,
 const Tensor& result) {
  if (result.numel() == 0) {
    return;
  }

  auto materialized = tensors.materialize();

  // We parallelize the copy if all 6 conditions pass:
  //
  // 1. There is more than one input tensor
  // 2. The out tensor is 32-bit indexable
  // 3. The number of dimensions is <= 4
  // 4. All input tensors are contiguous (output tensor may be non-contig)
  // 5. All input tensors can use 32-bit indexing

  const bool all32BitIndexable = std::all_of(materialized.begin(), materialized.end(),
    [] (const Tensor& t) {
      return at::cuda::detail::canUse32BitIndexMath(t);
    });

  int nDims = materialized[valid].get().dim();

  // We support the contiguous inputs and non-contiguous input (<=4 dims) in different ways
  // For contiguous input, we don't need to pass stride meta data to cuda kernel through constant
  // memory. Therefore, we could pass more inputs to cuda threads.
  // For non-contiguous, we reduce the number of inputs passed to cuda kernel due to the limitation
  // of constant memory.



  if (materialized.size() > 1 &&
      result.dim() <= CAT_ARRAY_MAX_INPUT_DIMS &&
      at::cuda::detail::canUse32BitIndexMath(result) &&
      all_contiguous &&
      all32BitIndexable &&
      all_same_dtype) {
      if (isBitsType(result.scalar_type())) {
        AT_DISPATCH_BIT_TYPES(result.scalar_type(), "cat_cuda", [&]() {
          using dtype = OpaqueType<sizeof(scalar_t)>;
          parallel_cat<dtype, CAT_ARRAY_BATCH_SIZE, 1>(result, materialized, dim, nDims, memory_format);
        });
      } else {
        AT_DISPATCH_V2(
            result.scalar_type(),
            "cat_cuda",
            AT_WRAP([&]() {
              using dtype = OpaqueType<sizeof(scalar_t)>;
              parallel_cat<dtype, CAT_ARRAY_BATCH_SIZE, 1>(
                  result, materialized, dim, nDims, memory_format);
            }),
            AT_EXPAND(AT_ALL_TYPES_AND_COMPLEX),
            kComplexHalf,
            kHalf,
            kBool,
            kBFloat16,
            AT_EXPAND(AT_FLOAT8_TYPES),
            AT_EXPAND(AT_BAREBONES_UNSIGNED_TYPES));
      }
  } else if (materialized.size() > 1 &&
      result.dim() <= CAT_ARRAY_MAX_INPUT_DIMS &&
      at::cuda::detail::canUse32BitIndexMath(result) &&
      nDims <= CAT_ARRAY_MAX_INPUT_DIMS &&
      all32BitIndexable &&
      all_same_dtype &&
      memory_format == c10::MemoryFormat::Contiguous) {
      if (isBitsType(result.scalar_type())) {
        AT_DISPATCH_BIT_TYPES(result.scalar_type(), "cat_cuda", [&]() {
          using dtype = OpaqueType<sizeof(scalar_t)>;
          parallel_cat<dtype, CAT_ARRAY_BATCH_SIZE/2, CAT_ARRAY_BATCH_SIZE/2>(result, materialized, dim, nDims, memory_format);
        });
      } else {
        AT_DISPATCH_V2(
            result.scalar_type(),
            "cat_cuda",
            AT_WRAP([&]() {
              using dtype = OpaqueType<sizeof(scalar_t)>;
              parallel_cat<
                  dtype,
                  CAT_ARRAY_BATCH_SIZE / 2,
                  CAT_ARRAY_BATCH_SIZE / 2>(
                  result, materialized, dim, nDims, memory_format);
            }),
            AT_EXPAND(AT_ALL_TYPES_AND_COMPLEX),
            kComplexHalf,
            kHalf,
            kBool,
            kBFloat16,
            kFloat8_e4m3fn,
            kFloat8_e4m3fnuz,
            kFloat8_e5m2,
            kFloat8_e5m2fnuz,
            AT_EXPAND(AT_BAREBONES_UNSIGNED_TYPES));
      }
  } else {
    int64_t offset = 0;
    for (const Tensor& t : materialized) {
      if (cat_should_skip_tensor(t)) continue;
      int64_t dimSize = t.size(dim);
      Tensor nt = at::narrow(result, dim, offset, dimSize);
      copy_(nt, t);
      offset += dimSize;
    }
  }
}

} // namespace at::native
